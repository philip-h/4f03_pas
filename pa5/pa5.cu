#include "hip/hip_runtime.h"
/*
 * Authors: Philip Habib
 *          Theo Stone
 * Date: April 25, 2017
 *
 * This program will blur a given image in ppm format with a given blur radius. It uses
 * the CUDA framework to blur the image with the GPU, giving us access to many threads.
 * The bulrPPM function will be called from the 'host' (CPU) and will run on the 
 * 'device' (GPU).
 *
 * Blur funciton is quite simple - the color of a pixel is determined by averaging the colors 
 * of all pixels around it, blur radius pixels from the current pixel.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include "ppmFile.cuh"

const char usage [] = "./pa5.x r <inputFilename>.ppm <outputFilename>.ppm\n"
      "where:\n"
      "r is the bulrr radians in pixels\n"
      "<inputFilename>.ppm is the name of the file to blur\n"
      "<outputFilename>.ppm is the name of the file to save the blurred image\n\n\n";

/*
 * blurPPM implements a simple blur algorithm: color of pixel is the average of all pixels around this pixel.
 * This function is called from the CPU and is run on the GPU (this is what the __global__ decorator means.
 * The blurPPM function itself only blurs one pixel [@index]. Will be run in parallel to eventually blur every pixel in an image.
 *
 * Input
 * unsigned char *dataIn - pointer to data array for Image struct of the input image
 * int *radius - blur radius. Defines how many pixels away from the current pixel we need to calculate the color of the current pixel.
 * int *width - The width of the image. Used to determine the (x,y) coordinate of the image
 * int *height - The height of the image. Also used to determine the (x,y) coordinate of the image
 *
 * Output
 * unsigned char *dataOut - pointer to data array for Image struct of the output image
 */
__global__ void blurPPM(unsigned char *dataIn, unsigned char *dataOut, int *radius, int *width, int *height) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int totalR = 0, totalG = 0, totalB = 0, numPixels = 0;

  // x,y of current position of the pixel given by index
  int x = (int)(index % (*width) );
  int y = (int)(index / (*width) );


  /* **************** Average Pixel **************** */
  /* Calculates the color of the current pixel */
  for(int bY = y-*radius; bY <= y+*radius; bY++)
  {
    for(int bX = x-*radius; bX <= x + *radius;  bX++)
    {
      if(bY < 0 || bX < 0 || bX > *width || bY > *height)
      {
        continue;
      }
      else
      {
        int offset = (bY*(*width) + bX)*3;
        /*  
          0 - RED channel
          1 - GREEN channel
          2 - BLUE channel
        */
        totalR += dataIn[offset + 0];
        totalG += dataIn[offset + 1];
        totalB += dataIn[offset + 2];
        numPixels ++;
      }
    }
  }

  int newR = (int)(totalR / numPixels);
  int newG = (int)(totalG / numPixels);
  int newB = (int)(totalB / numPixels);
  /* ********************************* */

  /*  
    0 - RED channel
    1 - GREEN channel
    2 - BLUE channel
  */

  int offset = (y*(*width) + x)*3;
  dataOut[offest + 0] = newR;
  dataOut[offset + 1] = newG;
  dataOut[offset + 2] = newB;

  
}
/*
 * The main function. Handles command line arguments and represents the host program.
 *
 * Input:
 * int argc - Number of command line arguments
 * char** argv - The command line arguments 
 */
int main(int argc, char** argv) {
  int radius;
  Image *imgIn, *imgOut;

  // Remove the filename from the argv list
  argv ++;
  argc --;

  if (argc < 3) {
      fprintf(stderr, "Invalid number of arguments. Usage:\n%s", usage);
      return 1;
  }

  radius = atoi(argv[0]);

  if (radius == 0) {
    fprintf(stderr, "No need to blur - radius is 0\n\n\n");
    return 1;
  }

  int width, height, size;
  imgIn = ImageRead(argv[1]);
  width = ImageWidth(imgIn);
  height = ImageHeight(imgIn);
  size = width * height * 3;

  imgOut = ImageCreate(width, height);

  // Timing with cuda events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  // Initialize device vars 
  unsigned char *dev_dataIn, *dev_dataOut;
  int *dev_radius, *dev_width, *dev_height;
  hipMalloc((void**) &dev_dataIn,  size);
  hipMalloc((void**) &dev_dataOut, size);
  hipMalloc((void**) &dev_radius,  sizeof(int));
  hipMalloc((void**) &dev_width,   sizeof(int));
  hipMalloc((void**) &dev_height,   sizeof(int));

  // Copy device vars to GPU
  hipMemcpy(dev_dataIn,  imgIn->data, size,  hipMemcpyHostToDevice );
  hipMemcpy(dev_radius,  &radius, sizeof(int),  hipMemcpyHostToDevice );
  hipMemcpy(dev_width,  &width, sizeof(int),  hipMemcpyHostToDevice );
  hipMemcpy(dev_height,  &height, sizeof(int),  hipMemcpyHostToDevice );

  hipEventRecord(start);
  blurPPM<<< (width*height)/1024  , 1024 >>>(dev_dataIn, dev_dataOut, dev_radius, dev_width, dev_height);
  hipEventRecord(stop);

  // Copy output from blurPPM to local data array from Image struct
  hipMemcpy(imgOut->data, dev_dataOut, size, hipMemcpyDeviceToHost );

  ImageWrite(imgOut, argv[2]);

  // Calculate the time it took blurPPM to run
  hipEventSynchronize(stop);
  float milis = 0;
  hipEventElapsedTime(&milis, start, stop);
  printf("Elapsed time %f \n", milis);

  hipFree(dev_dataIn);
  hipFree(dev_dataOut);
  hipFree(dev_radius);
  hipFree(dev_width);
  hipFree(dev_height);
  
}


