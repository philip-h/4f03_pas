#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include "ppmFile.cuh"

/* Globals */
// const int MAX_STRING = 100;
// const int RED = 0;
// const int GREEN = 1;
// const int BLUE = 2;

const char usage [] = "./pa5.x r <inputFilename>.ppm <outputFilename>.ppm\n"
      "where:\n"
      "r is the bulrr radians in pixels\n"
      "<inputFilename>.ppm is the name of the file to blur\n"
      "<outputFilename>.ppm is the name of the file to save the blurred image\n\n\n";

__global__ void blurPPM(unsigned char *dataIn, unsigned char *dataOut) {
  dataOut[threadIdx.x] = dataIn[threadIdx.x];
}

int main(int argc, char** argv) {
  /* Process command line arguments */
  int radius;
  Image *imgIn, *imgOut;

  // Remove the filename from the argv list
  argv ++;
  argc --;

  if (argc < 3) {
      fprintf(stderr, "Invalid number of arguments. Usage:\n%s", usage);
      return 1;
  }

  radius = atoi(argv[0]);

  if (radius == 0) {
    fprintf(stderr, "No need to blur - radius is 0\n\n\n");
    return 1;
  }

  int width, height, size;
  imgIn = ImageRead(argv[1]);
  width = ImageWidth(imgIn);
  height = ImageHeight(imgIn);
  size = width * height * 3;

  imgOut = ImageCreate(width, height);

  // Device Vars 
  unsigned char *dev_dataIn, *dev_dataOut;
  hipMalloc((void**) &dev_dataIn,  size);
  hipMalloc((void**) &dev_dataOut, size);

  hipMemcpy(dev_dataIn,  imgIn->data, size,  hipMemcpyHostToDevice );

  blurPPM<<< size , 1 >>>(dev_dataIn, dev_dataOut);

  hipMemcpy(imgOut->data, dev_dataOut, size, hipMemcpyDeviceToHost );
  
  ImageWrite(imgOut, argv[2]);
  
}


