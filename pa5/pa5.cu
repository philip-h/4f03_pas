#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include "ppmFile.cuh"

/* Globals */
const char usage [] = "./pa5.x r <inputFilename>.ppm <outputFilename>.ppm\n"
      "where:\n"
      "r is the bulrr radians in pixels\n"
      "<inputFilename>.ppm is the name of the file to blur\n"
      "<outputFilename>.ppm is the name of the file to save the blurred image\n\n\n";

__global__ void blurPPM(unsigned char *dataIn, unsigned char *dataOut, int *radius, int *width, int *height) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int totalR = 0, totalG = 0, totalB = 0, numPixels = 0;

  // x,y of current position given by index
  int x = (int)(index % (*width) );
  int y = (int)(index / (*width) );

  for(int bY = y-*radius; bY <= y+*radius; bY++)
  {
    for(int bX = x-*radius; bX <= x + *radius;  bX++)
    {
      if(bY < 0 || bX < 0 || bX > *width || bY > *height)
      {
        continue;
      }
      else
      {
        totalR += dataIn[(bY*(*width) + bX)*3 + 0];
        totalG += dataIn[(bY*(*width) + bX)*3 + 1];
        totalB += dataIn[(bY*(*width) + bX)*3 + 2];
        numPixels ++;
      }
    }
  }

  int newR = (int)(totalR / numPixels);
  int newG = (int)(totalG / numPixels);
  int newB = (int)(totalB / numPixels);

  dataOut[(y*(*width) + x)*3 + 0] = newR;
  dataOut[(y*(*width) + x)*3 + 1] = newG;
  dataOut[(y*(*width) + x)*3 + 2] = newB;

  
}

int main(int argc, char** argv) {
  /* Process command line arguments */
  int radius;
  Image *imgIn, *imgOut;

  // Remove the filename from the argv list
  argv ++;
  argc --;

  if (argc < 3) {
      fprintf(stderr, "Invalid number of arguments. Usage:\n%s", usage);
      return 1;
  }

  radius = atoi(argv[0]);

  if (radius == 0) {
    fprintf(stderr, "No need to blur - radius is 0\n\n\n");
    return 1;
  }

  int width, height, size;
  imgIn = ImageRead(argv[1]);
  width = ImageWidth(imgIn);
  height = ImageHeight(imgIn);
  size = width * height * 3;
  printf("size: %d \n", size);

  imgOut = ImageCreate(width, height);

  // Device Vars 
  unsigned char *dev_dataIn, *dev_dataOut;
  int *dev_radius, *dev_width, *dev_height;
  hipMalloc((void**) &dev_dataIn,  size);
  hipMalloc((void**) &dev_dataOut, size);
  hipMalloc((void**) &dev_radius,  sizeof(int));
  hipMalloc((void**) &dev_width,   sizeof(int));
  hipMalloc((void**) &dev_height,   sizeof(int));

  hipMemcpy(dev_dataIn,  imgIn->data, size,  hipMemcpyHostToDevice );
  hipMemcpy(dev_radius,  &radius, sizeof(int),  hipMemcpyHostToDevice );
  hipMemcpy(dev_width,  &width, sizeof(int),  hipMemcpyHostToDevice );
  hipMemcpy(dev_height,  &height, sizeof(int),  hipMemcpyHostToDevice );

  blurPPM<<< (width*height)/1024  , 1024 >>>(dev_dataIn, dev_dataOut, dev_radius, dev_width, dev_height);

  hipMemcpy(imgOut->data, dev_dataOut, size, hipMemcpyDeviceToHost );
  
  ImageWrite(imgOut, argv[2]);

  hipFree(dev_dataIn);
  hipFree(dev_dataOut);
  hipFree(dev_radius);
  hipFree(dev_width);
  hipFree(dev_height);
  
}


